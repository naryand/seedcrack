// diamond ore CUDA cracker

#include <hip/hip_runtime.h>
#include <stdio.h>

// unsigned long long typedef
typedef unsigned long long u64;

// sets internal seed
__device__ void set_seed(u64 *seed, u64 set) {
    *seed = set; *seed = (*seed ^ 0x5DEECE66DL) & ((1L << 48) - 1);
}

// gets the top n bits off the internal seed
__device__ int next(u64 *seed, int n) {
    *seed = (*seed * 0x5DEECE66DL + 0xBL) & ((1L << 48) - 1);
    return (int) (*seed >> (48 - n));
}

// returns a pseudorandom int given a seed
__device__ int next_int(u64 *seed) { return next(seed, 32); }

// returns bounded pseudorandom int given a seed and bound n
__device__ int next_int_bound(u64 *seed, int n) {
    if(n <= 0) { return -1; }
    if((n & -n) == n) { return (int) ((n * (long) next(seed, 31)) >> 31); } // i.e., n is next power of 2

    int bits, val;
    do { 
        bits = next(seed, 31); 
        val = bits % n; 
    } while(bits - val + (n - 1) < 0);
    return val;
}

// returns pseudorandom long long given a seed
__device__ u64 next_long(u64 *seed) { return (((long long) next(seed, 32)) << 32) + (long long) next(seed, 32); }

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        exit(code);
    }
}

typedef struct ore { char chunk_x; char chunk_z; char x; char y; char z; } ore;

// ores step 6, diamond index 9, emerald index 14
__device__ void set_decorator_seed(u64 *seed, u64 world_seed, int x, int z, char index, char step) {
    set_seed(seed, world_seed);
    u64 a = next_long(seed) | 1L;
    u64 b = next_long(seed) | 1L;
    u64 population_seed = (((u64) x)*a+((u64) z)*b)^world_seed;
    u64 decorator_seed = population_seed + (index + (step*10000)); // bottom max 30 bits affects parity
    set_seed(seed, decorator_seed);
}

// 82449439703029
#define COUNT 4
__constant__ ore arr[COUNT] = {{0,0, 1,2,4},
                               {0,1, 8,6,15},
                               {0,2, 2,2,6},
                               {1,0, 1,8,5}};

__global__ void find(int grid, u64 *flag) {
    u64 seed[1]; 
    // calculate block
    u64 i = grid*gridDim.x+blockIdx.x;
    // calculate thread
    i = i*blockDim.x+threadIdx.x;
    // printf("%llu\n", i);
    for(char j = 0; j < COUNT; j++) { 
        set_decorator_seed(seed, i, arr[j].chunk_x << 4, arr[j].chunk_z << 4, 9, 6);
        char x = next(seed, 4);
        char z = next(seed, 4);
        char y = next(seed, 4);
        // printf("%d %d %d != %d %d %d\n", x,z,y, arr[j].x, arr[j].z, arr[j].y);
        if(x != arr[j].x) { return; }
        if(z != arr[j].z) { return; }
        if(y != arr[j].y) { return; }
    }
    printf("found seed %llu\n", i);
    // printf("%d %d %d %d %d %d\n", grid, gridDim.x, blockIdx.x, blockDim.x, threadIdx.x, i);
}

#define GRIDS 1<<8
#define BLOCKS 1<<30

#define THREADS 1<<10

int main() {
    u64 *flag = 0;
    hipMallocManaged(&flag, sizeof(u64));
    for(int i = 0; i < GRIDS; i++) {
        find<<<BLOCKS,THREADS>>>(i, flag);
        GPU_ASSERT(hipDeviceSynchronize());
        GPU_ASSERT(hipPeekAtLastError());
    }
}